#include "hip/hip_runtime.h"


#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define CLD(N, D) ((N + D - 1) / D)

__global__ void device_compute_edge_offsets(
    int64_t * dsts, // [NE]
    int64_t * out, // [NN]
    int NE,
    int NN
) {
    const int e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= NE) return;
    atomicAdd((unsigned long long *)&out[dsts[e]], (unsigned long long)1);
}

at::Tensor compute_edge_offsets(
    at::Tensor dsts, // [NE]
    int64_t NN
) {
    CHECK_INPUT(dsts);

    int64_t NE = dsts.size(0);
    at::Tensor out = torch::zeros({NN}, dsts.options());

    device_compute_edge_offsets<<<CLD(NE, 512), 512>>>(
        dsts.data_ptr<int64_t>(),
        out.data_ptr<int64_t>(),
        (int)NE,
        (int)NN
    );

    return out.cumsum(0);
}


template<typename scalar_t, size_t MAX_D>
__global__ void device_fused_gather_concat_2e(
    const scalar_t * nf, // [N, D]
    const scalar_t * ef0, const int64_t * ef0_offsets, // [N, D], [N]
    const scalar_t * ef1, const int64_t * ef1_offsets, // [N, D], [N]
    scalar_t * out, // [N, 3D]
    const int64_t N,
    const int64_t NE0,
    const int64_t NE1,
    const int64_t D
) {
    __shared__ scalar_t accum[MAX_D];

    const int NES = NE1 == 0 ? 1 : 2;
    const int node_i = blockIdx.x;
    const int node_d = threadIdx.x;
    const int row_off = node_i * D;
    const int out_row_off = node_i * (NES + 1) * D;
    const int d_accum_off = node_d - D;

    assert(d_accum_off < (int)MAX_D);
    assert(node_i < N);

    if (node_d < D) {
        out[out_row_off + node_d] = nf[row_off + node_d];
    } else if (node_d < 2*D) {
        const int d_off = node_d - D;
        accum[d_accum_off] = (scalar_t)(0.0f);

        const int e_start = (node_i == 0) ? 0 : ef0_offsets[node_i - 1];
        const int e_end = ef0_offsets[node_i];

        for (int e = e_start; e < e_end; e++) {
            accum[d_accum_off] = (accum[d_accum_off] + ef0[e * D + d_off]);
        }

        out[out_row_off + node_d] = accum[d_accum_off];
    } else {
        const int d_off = node_d - D - D;
        accum[d_accum_off] = (scalar_t)(0.0f);

        const int e_start = (node_i == 0) ? 0 : ef1_offsets[node_i - 1];
        const int e_end = ef1_offsets[node_i];

        for (int e = e_start; e < e_end; e++) {
            accum[d_accum_off] = (accum[d_accum_off] + ef1[e * D + d_off]);
        }

        out[out_row_off + node_d] = accum[d_accum_off];
    }
}

at::Tensor fused_gather_concat_2e(
    at::Tensor nf,
    at::Tensor ef0,
    at::Tensor eoffs0,
    at::Tensor ef1,
    at::Tensor eoffs1
) {
    CHECK_INPUT(nf);
    CHECK_INPUT(ef0);
    CHECK_INPUT(eoffs0);
    CHECK_INPUT(ef1);
    CHECK_INPUT(eoffs1);

    const int64_t D = nf.size(1);
    const int64_t NN = nf.size(0);
    const int64_t NE0 = ef0.size(0);
    const int64_t NE1 = ef1.size(0);

    assert(D <= 128);
    assert(ef0.size(1) == D);
    assert(ef1.size(1) == D);

    at::Tensor out = at::zeros({NN, 3*D}, nf.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(nf.scalar_type(), "device_fused_gather_concat_2e", [&] {
        device_fused_gather_concat_2e<scalar_t, 256><<<NN, 3*D>>>(
            (scalar_t *)nf.data_ptr<scalar_t>(),
            (scalar_t *)ef0.data_ptr<scalar_t>(), eoffs0.data_ptr<int64_t>(),
            (scalar_t *)ef1.data_ptr<scalar_t>(), eoffs1.data_ptr<int64_t>(),
            (scalar_t *)out.data_ptr<scalar_t>(),
            NN,
            NE0,
            NE1,
            D
        );
    });

    return out;
}


at::Tensor fused_gather_concat_1e(
    at::Tensor nf,
    at::Tensor ef0,
    at::Tensor eoffs0
) {
    CHECK_INPUT(nf);
    CHECK_INPUT(ef0);
    CHECK_INPUT(eoffs0);

    const int64_t D = nf.size(1);
    const int64_t NN = nf.size(0);
    const int64_t NE0 = ef0.size(0);

    assert(D <= 128);
    assert(ef0.size(1) == D);

    at::Tensor out = at::zeros({NN, 2*D}, nf.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(nf.scalar_type(), "device_fused_gather_concat_2e", [&] {
        device_fused_gather_concat_2e<scalar_t, 128><<<NN, 2*D>>>(
            (scalar_t *)nf.data_ptr<scalar_t>(),
            (scalar_t *)ef0.data_ptr<scalar_t>(), eoffs0.data_ptr<int64_t>(),
            nullptr, nullptr,
            (scalar_t *)out.data_ptr<scalar_t>(),
            NN,
            NE0,
            0,
            D
        );
    });

    return out;
}


at::Tensor test(std::vector<at::Tensor> x) {
    return torch::zeros({1});
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("test", &test, "Test");
    m.def("compute_edge_offsets", &compute_edge_offsets, "Compute Edge Offsets");
    m.def("fused_gather_concat_1e", &fused_gather_concat_1e, "Fused Gather Concat 1e");
    m.def("fused_gather_concat_2e", &fused_gather_concat_2e, "Fused Gather Concat 2e");
}
