#include "hip/hip_runtime.h"


#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define CLD(N, D) ((N + D - 1) / D)

__global__ void device_fused_scatter_concat(
    half * ef, // [NE, D]
    half * nf, // [NN, D]
    half * out, // [NE, 3*D]
    int64_t * srcs, // [NE]
    int64_t * dsts, // [NE]
    const int64_t NN,
    const int64_t NE,
    const int64_t D
) {
    const int edge_i = blockIdx.x;
    const int edge_d = threadIdx.x;

    if (edge_d < D) {
        out[edge_i * 3 * D + edge_d] = ef[edge_i * D + edge_d];
    } else if (edge_d < 2 * D) {
        out[edge_i * 3 * D + edge_d] = nf[srcs[edge_i] * D + edge_d - D];
    } else {
        out[edge_i * 3 * D + edge_d] = nf[dsts[edge_i] * D + edge_d - D - D];
    }

}

void fused_scatter_concat_out(
    at::Tensor ef,
    at::Tensor nf,
    at::Tensor srcs,
    at::Tensor dsts,
    at::Tensor out
) {
    CHECK_INPUT(ef);
    CHECK_INPUT(nf);
    CHECK_INPUT(out);
    CHECK_INPUT(srcs);
    CHECK_INPUT(dsts);

    const int64_t D = ef.size(1);
    const int64_t NN = nf.size(0);
    const int64_t NE = ef.size(0);

    assert(D <= 128);
    assert(ef.size(1) == D);

    device_fused_scatter_concat<<<NE, 3*D>>>(
        (half *)ef.data_ptr<at::Half>(),
        (half *)nf.data_ptr<at::Half>(),
        (half *)out.data_ptr<at::Half>(),
        srcs.data_ptr<int64_t>(),
        dsts.data_ptr<int64_t>(),
        NN,
        NE,
        D
    );
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_scatter_concat_out", &fused_scatter_concat_out, "Fused Scatter Concat Out");
}
